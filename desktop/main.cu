#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <cmath>
#include <thread>
#include <iomanip>
#include <string>
#include <cassert>
#include <cstring>
#include <stdio.h>

#include "sha256.cuh"

#define SHOW_INTERVAL_MS 2000
//#define BLOCK_SIZE 256
#define SHA_PER_ITERATIONS 8'388'608
//#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE
#define BLOCK_SIZE 4
#define NUMBLOCKS 4

static size_t difficulty = 1;

// Output string by the device read by host
unsigned char *g_hash_out = nullptr;
int *g_found = nullptr;
int *g_foundval = nullptr;

static uint64_t nonce = 0;
static uint64_t user_nonce = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug infos
static std::chrono::high_resolution_clock::time_point t_last_updated;

__device__ bool checkZeroPadding(unsigned char *sha, uint8_t difficulty)
{

    bool isOdd = difficulty % 2 != 0;
    uint8_t max = (difficulty / 2) + 1;

    /*
        Odd : 00 00 01 need to check 0 -> 2
        Even : 00 00 00 1 need to check 0 -> 3
        odd : 5 / 2 = 2 => 2 + 1 = 3
        even : 6 / 2 = 3 => 3 + 1 = 4
    */
    for (uint8_t cur_byte = 0; cur_byte < max; ++cur_byte)
    {
        uint8_t b = sha[cur_byte];
        if (cur_byte < max - 1)
        { // Before the last byte should be all zero
            if (b != 0)
                return false;
        }
        else if (isOdd)
        {
            if (b > 0x0F || b == 0)
                return false;
        }
        else if (b <= 0x0f)
            return false;
    }

    return true;
}

// Does the same as sprintf(char*, "%d%s", int, const char*) but a bit faster
__device__ uint8_t nonce_to_str(uint64_t nonce, unsigned char *out)
{
    uint64_t result = nonce;
    uint8_t remainder;
    uint8_t nonce_size = nonce == 0 ? 1 : floor(log10((double)nonce)) + 1;
    uint8_t i = nonce_size;
    while (result >= 10)
    {
        remainder = result % 10;
        result /= 10;
        out[--i] = remainder + '0';
    }

    out[0] = result + '0';
    i = nonce_size;
    out[i] = 0;
    return i;
}

struct Block
{
    char *blockid;
    char *prevhash;
    int difficulty;
    // int proof; //our implementation will directly inject proof value
    char *transactions; // JSON array
};

__device__ int my_strlen(char *src)
{
    int i = 0;
    while (src[i++] != 0)
        ;
    return i-1;
}

__device__ char *my_strcpy(char *dest, const char *src)
{
    int i = 0;
    do
    {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src)
{
    int i = 0;
    while (dest[i] != 0)
        i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ void reverse(char str[], int length)
{
    int start = 0;
    int end = length - 1;
    while (start < end)
    {
        char tmp = *(str + start);
        *(str + start) = *(str + end);
        *(str + end) = tmp;
        start++;
        end--;
    }
}

__device__ char *itoa(uint64_t num, char *str)
{
    int i = 0;
    if (num == 0)
    {
        str[i++] = '0';
        str[i] = '\0';
        return str;
    }
    while (num != 0)
    {
        int rem = num % 10;
        str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
        num = num / 10;
    }
    str[i] = '\0';
    reverse(str, i);
    return str;
}

__device__ char *formJSONStr(char *dest, char *blockid, char *prevhash, char *transactions, uint8_t difficulty, uint64_t nonce)
{
    /*
    blockid,
    difficulty,
    prevhash,
    proof,
    transactions
    */
    /*printf("Difficulty: %d\n", difficulty);
    printf("Blockid: %s\n", blockid);
    printf("So far: %s\n", dest);*/
    my_strcat(dest, "{\"blockid\":\"");
    my_strcat(dest, blockid);
    my_strcat(dest, "\",\"difficulty\":");
    // printf("So far: %s\n", dest);
    char diff[32];
    my_strcat(dest, itoa(difficulty, diff));
    my_strcat(dest, ",\"prevhash\":\"");
    my_strcat(dest, prevhash);
    my_strcat(dest, "\",\"proof\":");
    // printf("So far: %s\n", dest);
    char noncestr[32];
    my_strcat(dest, itoa(nonce, noncestr));
    my_strcat(dest, ",\"transactions\":");
    my_strcat(dest, transactions);
    my_strcat(dest, "}");
    // printf("So far: %s\n", dest);
    return dest;
}

extern __shared__ char array[];
__global__ void sha256_kernel(unsigned char *out_found_hash, int *out_found, int *out_foundval, const char *in_blockid, const char *in_prevhash, const char *in_transactions, size_t in_bidsize, size_t in_bphashsize, size_t in_btranssize, size_t in_block_size, uint8_t difficulty, uint64_t nonce_offset)
{

    // If this is the first thread of the block, init the input block in shared memory
    // Block *in = (Block*) &blockdata;
    // printf("owo1\n");

    char *in_bid = (char *)&array[0];
    char *in_bphash = (char *)&array[in_bidsize + 1];
    char *in_btrans = (char *)&array[in_bidsize + in_bphashsize + 2];

    if (threadIdx.x == 0)
    {
        // printf("%p %d ", in, sizeof(blockdata));
        // printf("%p %d ", in_block, in_block_size);
        // printf("%s %s %s %d ", in_block->blockid, in_block->prevhash, in_block->transactions, in_block->difficulty);
        // memcpy(in, in_block, in_block_size);
        // printf("%s %s %s\n", in_blockid, in_prevhash, in_transactions);
        // printf("%d %d %d\n", in_bidsize, in_bphashsize, in_btranssize);
        /*memcpy(in_bid, in_blockid, in_bidsize + 1);
        memcpy(in_bphash, in_prevhash, in_bphashsize + 1);
        memcpy(in_btrans, in_transactions, in_btranssize + 1);*/
        my_strcpy(in_bid, in_blockid);
        my_strcpy(in_bphash, in_prevhash);
        my_strcpy(in_btrans, in_transactions);
        // printf("%s %s %s\n", in_bid, in_bphash, in_btrans);
        /*in->blockid = in_blockid;
        in->prevhash = in_prevhash;
        in->transactions = in_transactions;*/
        // in->difficulty = difficulty;
        // printf("owo3");
        // printf("\n");
        // printf("Copied blockid: %s", in_bid);
    }

    // printf("owo2");

    __syncthreads(); // Ensure the input block has been written in SMEM

    // printf("owo\n");

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t nonce = idx + nonce_offset;

    // The first byte we can write because there is the input string at the begining
    // Respects the memory padding of 8 bit (char).
    size_t const minArray = static_cast<size_t>(ceil((in_block_size + 3) / 8.f) * 8);

    uintptr_t sha_addr = threadIdx.x * (64) + minArray;
    uintptr_t nonce_addr = sha_addr + 32;

    // unsigned char *sha = (unsigned char *)&array[sha_addr];
    unsigned char *out = (unsigned char *)&array[nonce_addr];
    // memset(out, 0, 32);

    char *sha_tmp = (char *)malloc(sizeof(char) * (32));
    sha_tmp[0] = '\0';
    unsigned char *sha = (unsigned char *)sha_tmp;

    uint8_t size = nonce_to_str(nonce, out);

    assert(size <= 32);

    // printf("owo?\n");

    {
        // unsigned char tmp[32];

        SHA256_CTX ctx;
        sha256_init(&ctx);
        // printf("owo2?\n");
        //  sha256_update(&ctx, out, size);
        char *dest = (char *)malloc(sizeof(char) * (1));
        dest[0] = '\0';
        // printf("owoo!\n");
        //printf("%d blockidx %d %d %d blockdim %d %d %d threadidx %d %d %d\n", nonce, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.y, blockDim.z, threadIdx.x, threadIdx.y, threadIdx.z);
        formJSONStr(dest, in_bid, in_bphash, in_btrans, difficulty, nonce);
        //printf("%s %d\n", dest, sizeof(char) * my_strlen(dest));
        // error here (todo: fix)
        // printf("owo3?\n");
        sha256_update(&ctx, (unsigned char *)dest, sizeof(char) * my_strlen(dest));
        // printf("owo4?\n");
        free(dest);
        sha256_final(&ctx, sha);

        // Second round of SHA256
        /*sha256_init(&ctx);
        sha256_update(&ctx, tmp, 32);
        sha256_final(&ctx, sha);*/
    }
    // printf("owo5?\n");

    //printf("%s\n", sha);

    if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0 && atomicExch(out_foundval, nonce) == 0)
    { // if zero padding, checks if subbing *out_found with 1 is successful
        memcpy(out_found_hash, sha, 32);
    }
    free(sha_tmp);
}

void pre_sha256()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char *sha256)
{
    for (uint8_t i = 0; i < 32; ++i)
    {
        std::cout << std::hex << std::setfill('0') << std::setw(2) << static_cast<int>(sha256[i]);
    }
    std::cout << std::dec << std::endl;
}

void print_state()
{
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> last_show_interval = t2 - t_last_updated;

    if (last_show_interval.count() > SHOW_INTERVAL_MS)
    {
        std::chrono::duration<double, std::milli> span = t2 - t_last_updated;
        float ratio = span.count() / 1000;
        std::cout << span.count() << " " << nonce - last_nonce_since_update << std::endl;

        std::cout << std::fixed << static_cast<uint64_t>((nonce - last_nonce_since_update) / ratio) << " hash(es)/s" << std::endl;

        std::cout << std::fixed << "Nonce : " << nonce << std::endl;

        t_last_updated = std::chrono::high_resolution_clock::now();
        last_nonce_since_update = nonce;
    }

    if (*g_found)
    {
        print_hash(g_hash_out);
        std::cout << std::fixed << "Final Nonce : " << *g_foundval << std::endl;
    }
}

int main()
{

    hipSetDevice(0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    t_last_updated = std::chrono::high_resolution_clock::now();

    std::string blockid;
    std::string prevhash;
    std::string transactions;
    std::cout << "Enter block ID : ";
    getline(std::cin, blockid);
    std::cout << "Enter previous block's hash : ";
    getline(std::cin, prevhash);
    std::cout << "Enter transactions JSON : ";
    getline(std::cin, transactions);
    std::cout << "Enter difficulty : ";
    std::cin >> difficulty;
    std::cout << std::endl;

    char *d_in_blockid = nullptr;
    hipMalloc(&d_in_blockid, blockid.size() + 1);
    hipMemcpy(d_in_blockid, blockid.c_str(), blockid.size() + 1, hipMemcpyHostToDevice);

    char *d_in_prevhash = nullptr;
    hipMalloc(&d_in_prevhash, prevhash.size() + 1);
    hipMemcpy(d_in_prevhash, prevhash.c_str(), prevhash.size() + 1, hipMemcpyHostToDevice);

    char *d_in_transactions = nullptr;
    hipMalloc(&d_in_transactions, transactions.size() + 1);
    hipMemcpy(d_in_transactions, transactions.c_str(), transactions.size() + 1, hipMemcpyHostToDevice);

    hipMallocManaged(&g_hash_out, 32);
    hipMallocManaged(&g_found, sizeof(int));
    *g_found = 0;
    hipMallocManaged(&g_foundval, sizeof(int));
    *g_foundval = 0;

    pre_sha256();

    const size_t totalInpSize = blockid.size() + prevhash.size() + transactions.size();

    std::cout << totalInpSize << " (inp size)\n";

    size_t dynamic_shared_size = (ceil((totalInpSize + 1 + 1 + 1) / 8.f) * 8) + (64 * BLOCK_SIZE);

    std::cout << "Shared memory is " << dynamic_shared_size / 1024 << "KB " << dynamic_shared_size << std::endl;

    int ctr = 0;

    while (!*g_found)
    {
        // todo: modify to pass block data
        sha256_kernel<<<NUMBLOCKS, BLOCK_SIZE, dynamic_shared_size>>>(g_hash_out, g_found, g_foundval, d_in_blockid, d_in_prevhash, d_in_transactions, blockid.size(), prevhash.size(), transactions.size(), totalInpSize, difficulty, nonce);
        hipError_t err1 = hipGetLastError();
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
            std::cout << "Iterations: " << ctr << " " << nonce << std::endl;
            std::cout << err << std::endl;
            throw std::runtime_error("Device error");
        }

        nonce += NUMBLOCKS * BLOCK_SIZE;

        print_state();
        ctr++;
    }

    //std::cout << &g_found << " " << *g_found << " " << g_found << "\n";

    hipFree(g_hash_out);
    hipFree(g_found);
    hipFree(g_foundval);

    hipFree(d_in_blockid);
    hipFree(d_in_prevhash);
    hipFree(d_in_transactions);

    hipDeviceReset();

    return 0;
}
