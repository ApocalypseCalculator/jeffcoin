#include "hip/hip_runtime.h"
//Jeffcoin miner

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <cmath>
#include <thread>
#include <iomanip>
#include <string>
#include <cassert>
#include <cstring>
#include <stdio.h>

#include "sha256.cuh"

#define SHOW_INTERVAL_MS 2000
#define SHA_PER_ITERATIONS 8'388'608
//#define BLOCK_SIZE 256
//#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE

//these hardcoded values work
//todo: fix to not 700 illegal mem access for larger values (only 700s for long input strings)
#define BLOCK_SIZE 128
#define NUMBLOCKS 128

static size_t difficulty = 1;

// Output string by the device read by host
unsigned char *g_hash_out = nullptr;
int *g_found = nullptr;
int *g_foundval = nullptr;

static uint64_t nonce = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug infos
static std::chrono::high_resolution_clock::time_point t_last_updated;

__device__ bool checkZeroPadding(unsigned char *sha, uint8_t difficulty)
{
    bool isOdd = difficulty % 2 != 0;
    uint8_t max = (difficulty / 2) + 1;
    /*
        Odd : 00 00 01 need to check 0 -> 2
        Even : 00 00 00 1 need to check 0 -> 3
        odd : 5 / 2 = 2 => 2 + 1 = 3
        even : 6 / 2 = 3 => 3 + 1 = 4
    */
    for (uint8_t cur_byte = 0; cur_byte < max; ++cur_byte)
    {
        uint8_t b = sha[cur_byte];
        if (cur_byte < max - 1)
        { // Before the last byte should be all zero
            if (b != 0)
                return false;
        }
        else if (isOdd)
        {
            if (b > 0x0F || b == 0)
                return false;
        }
        else if (b <= 0x0f)
            return false;
    }

    return true;
}

__device__ int my_strlen(char *src)
{
    int i = 0;
    while (src[i++] != 0)
        ;
    return i - 1;
}

__device__ char *my_strcpy(char *dest, const char *src)
{
    int i = 0;
    do
    {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src)
{
    int i = 0;
    while (dest[i] != 0)
        i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ void reverse(char str[], int length)
{
    int start = 0;
    int end = length - 1;
    while (start < end)
    {
        char tmp = *(str + start);
        *(str + start) = *(str + end);
        *(str + end) = tmp;
        start++;
        end--;
    }
}

__device__ char *itoa(uint64_t num, char *str)
{
    int i = 0;
    if (num == 0)
    {
        str[i++] = '0';
        str[i] = '\0';
        return str;
    }
    while (num != 0)
    {
        int rem = num % 10;
        str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
        num = num / 10;
    }
    str[i] = '\0';
    reverse(str, i);
    return str;
}

__device__ char *formJSONStr(char *dest, char *blockid, char *prevhash, char *transactions,  char *difficulty,  char *nonce)
{
    /*
    blockid, (block id str)
    difficulty, (itoa'd difficulty level)
    prevhash, (last hash hex str)
    proof, (itoa'd nonce val)
    transactions (JSON arr)
    */
    my_strcat(dest, "{\"blockid\":\"");
    my_strcat(dest, blockid);
    my_strcat(dest, "\",\"difficulty\":");
    my_strcat(dest, difficulty);
    my_strcat(dest, ",\"prevhash\":\"");
    my_strcat(dest, prevhash);
    my_strcat(dest, "\",\"proof\":");
    my_strcat(dest, nonce);
    my_strcat(dest, ",\"transactions\":");
    my_strcat(dest, transactions);
    my_strcat(dest, "}");
    return dest;
}

extern __shared__ char array[];
__global__ void sha256_kernel(unsigned char *out_found_hash, int *out_found, int *out_foundval, const char *in_blockid, const char *in_prevhash, const char *in_transactions, size_t in_bidsize, size_t in_bphashsize, size_t in_btranssize, size_t in_block_size, uint8_t difficulty, uint64_t nonce_offset)
{
    char *in_bid = (char *)&array[0]; //inputted block id is first in SMEM
    char *in_bphash = (char *)&array[in_bidsize + 1]; //fetch block prevhash from SMEM offset
    char *in_btrans = (char *)&array[in_bidsize + in_bphashsize + 2]; //fetch block transaction list from SMEM offset

    // If this is the first thread of the block, init the input block data in shared memory
    if (threadIdx.x == 0)
    {
        my_strcpy(in_bid, in_blockid);
        my_strcpy(in_bphash, in_prevhash);
        my_strcpy(in_btrans, in_transactions);
    }

    __syncthreads(); // Ensure the input block has been written in SMEM

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t nonce = idx + nonce_offset;

    char *sha_tmp = (char *)malloc(sizeof(char) * (32));
    sha_tmp[0] = '\0';
    unsigned char *sha = (unsigned char *)sha_tmp;

    {
        SHA256_CTX ctx;
        sha256_init(&ctx);

        char diff[32];
        itoa(difficulty, diff);
        char noncestr[32];
        itoa(nonce, noncestr);

        // very hacky allocation size calculation
        size_t destsize = my_strlen(in_bid) + my_strlen(in_bphash) + my_strlen(in_btrans) + my_strlen(diff) + my_strlen(noncestr) + 12 + 15 + 13 + 10 + 16 + 1;

        char *dest = (char *)malloc(sizeof(char) * (destsize));
        dest[0] = '\0'; //null byte

        //create JSON string to hash
        formJSONStr(dest, in_bid, in_bphash, in_btrans, diff, noncestr);

        sha256_update(&ctx, (unsigned char *)dest, sizeof(char) * destsize);

        free(dest);
        
        sha256_final(&ctx, sha);
    }

    if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0 && atomicExch(out_foundval, nonce) == 0)
    { // if zero padding, checks if subbing *out_found with 1 is successful
        memcpy(out_found_hash, sha, 32);
    }
    free(sha_tmp);
}

void pre_sha256()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char *sha256)
{
    for (uint8_t i = 0; i < 32; ++i)
    {
        std::cout << std::hex << std::setfill('0') << std::setw(2) << static_cast<int>(sha256[i]);
    }
    std::cout << std::dec << std::endl;
}

void print_state()
{
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> last_show_interval = t2 - t_last_updated;

    if (last_show_interval.count() > SHOW_INTERVAL_MS)
    {
        std::chrono::duration<double, std::milli> span = t2 - t_last_updated;
        float ratio = span.count() / 1000;

        std::cout << std::fixed << static_cast<uint64_t>((nonce - last_nonce_since_update) / ratio) << " hash(es)/s" << std::endl;

        std::cout << std::fixed << "Current Nonce : " << nonce << std::endl;

        t_last_updated = std::chrono::high_resolution_clock::now();
        last_nonce_since_update = nonce;
    }

    if (*g_found)
    {
        print_hash(g_hash_out);
        std::cout << std::fixed << "Final Nonce : " << *g_foundval << std::endl;
    }
}

int main(int argc, char* argv[])
{

    hipSetDevice(0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    t_last_updated = std::chrono::high_resolution_clock::now();

    std::string blockid;
    std::string prevhash;
    std::string transactions;
    if(argc < 5) {
        std::cout << "Invalid arguments" << std::endl;
        return 1;
    }
    //std::cout << argv[1] << "\n" << argv[2] << "\n" << argv[3] << "\n" << argv[4] << "\n";
    blockid.assign(argv[1]);
    prevhash.assign(argv[2]);
    transactions.assign(argv[3]);
    try {
        difficulty = std::stoi(argv[4]);
    }
    catch(std::exception const &e) {
        std::cout << "Invalid difficulty" << std::endl;
        return 1;
    }
    /*std::cout << "Enter block ID : ";
    getline(std::cin, blockid);
    std::cout << "Enter previous block's hash : ";
    getline(std::cin, prevhash);
    std::cout << "Enter transactions JSON : ";
    getline(std::cin, transactions);
    std::cout << "Enter difficulty : ";
    std::cin >> difficulty;
    std::cout << std::endl;*/

    char *d_in_blockid = nullptr;
    hipMalloc(&d_in_blockid, blockid.size() + 1);
    hipMemcpy(d_in_blockid, blockid.c_str(), blockid.size() + 1, hipMemcpyHostToDevice);

    char *d_in_prevhash = nullptr;
    hipMalloc(&d_in_prevhash, prevhash.size() + 1);
    hipMemcpy(d_in_prevhash, prevhash.c_str(), prevhash.size() + 1, hipMemcpyHostToDevice);

    char *d_in_transactions = nullptr;
    hipMalloc(&d_in_transactions, transactions.size() + 1);
    hipMemcpy(d_in_transactions, transactions.c_str(), transactions.size() + 1, hipMemcpyHostToDevice);

    hipMallocManaged(&g_hash_out, 32);
    hipMallocManaged(&g_found, sizeof(int));
    *g_found = 0;
    hipMallocManaged(&g_foundval, sizeof(int));
    *g_foundval = 0;

    pre_sha256();

    const size_t totalInpSize = blockid.size() + prevhash.size() + transactions.size();

    //std::cout << totalInpSize << " (inp size)\n";

    size_t dynamic_shared_size = (ceil((totalInpSize + 1 + 1 + 1) / 8.f) * 8) + (64 * BLOCK_SIZE);

    std::cout << "Shared memory is " << dynamic_shared_size / 1024 << "KB" << std::endl;

    while (!*g_found)
    {
        sha256_kernel<<<NUMBLOCKS, BLOCK_SIZE, dynamic_shared_size>>>(g_hash_out, g_found, g_foundval, d_in_blockid, d_in_prevhash, d_in_transactions, blockid.size(), prevhash.size(), transactions.size(), totalInpSize, difficulty, nonce);
        //hipError_t err1 = hipGetLastError();
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
            std::cout << err << std::endl;
            throw std::runtime_error("Device error");
        }

        nonce += NUMBLOCKS * BLOCK_SIZE;

        print_state();
    }

    hipFree(g_hash_out);
    hipFree(g_found);
    hipFree(g_foundval);

    hipFree(d_in_blockid);
    hipFree(d_in_prevhash);
    hipFree(d_in_transactions);

    hipDeviceReset();

    return 0;
}
