#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <cmath>
#include <thread>
#include <iomanip>
#include <string>
#include <cassert>
#include <cstring>

#include "sha256.cuh"

#define SHOW_INTERVAL_MS 2000
#define BLOCK_SIZE 256
#define SHA_PER_ITERATIONS 8'388'608
#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE

static size_t difficulty = 1;

// Output string by the device read by host
char *g_out = nullptr;
unsigned char *g_hash_out = nullptr;
int *g_found = nullptr;

static uint64_t nonce = 0;
static uint64_t user_nonce = 0;
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug infos
static std::chrono::high_resolution_clock::time_point t_last_updated;

__device__ bool checkZeroPadding(unsigned char *sha, uint8_t difficulty)
{

    bool isOdd = difficulty % 2 != 0;
    uint8_t max = (difficulty / 2) + 1;

    /*
        Odd : 00 00 01 need to check 0 -> 2
        Even : 00 00 00 1 need to check 0 -> 3
        odd : 5 / 2 = 2 => 2 + 1 = 3
        even : 6 / 2 = 3 => 3 + 1 = 4
    */
    for (uint8_t cur_byte = 0; cur_byte < max; ++cur_byte)
    {
        uint8_t b = sha[cur_byte];
        if (cur_byte < max - 1)
        { // Before the last byte should be all zero
            if (b != 0)
                return false;
        }
        else if (isOdd)
        {
            if (b > 0x0F || b == 0)
                return false;
        }
        else if (b <= 0x0f)
            return false;
    }

    return true;
}

// Does the same as sprintf(char*, "%d%s", int, const char*) but a bit faster
__device__ uint8_t nonce_to_str(uint64_t nonce, unsigned char *out)
{
    uint64_t result = nonce;
    uint8_t remainder;
    uint8_t nonce_size = nonce == 0 ? 1 : floor(log10((double)nonce)) + 1;
    uint8_t i = nonce_size;
    while (result >= 10)
    {
        remainder = result % 10;
        result /= 10;
        out[--i] = remainder + '0';
    }

    out[0] = result + '0';
    i = nonce_size;
    out[i] = 0;
    return i;
}

struct Block
{
    char *blockid;
    char *prevhash;
    int difficulty;
    // int proof; //our implementation will directly inject proof value
    char *transactions; // JSON array
};

__device__ char *my_strcpy(char *dest, const char *src)
{
    int i = 0;
    do
    {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src)
{
    int i = 0;
    while (dest[i] != 0)
        i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ void reverse(char str[], int length)
{
    int start = 0;
    int end = length - 1;
    while (start < end)
    {
        char tmp = *(str + start);
        *(str + start) = *(str + end);
        *(str + end) = tmp;
        start++;
        end--;
    }
}

__device__ char *itoa(uint64_t num, char *str)
{
    int i = 0;
    if (num == 0)
    {
        str[i++] = '0';
        str[i] = '\0';
        return str;
    }
    while (num != 0)
    {
        int rem = num % 10;
        str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
        num = num / 10;
    }
    str[i] = '\0';
    reverse(str, i);
    return str;
}

__device__ char *formJSONStr(char *dest, Block *block, uint64_t nonce)
{
    /*
    blockid,
    difficulty,
    prevhash,
    proof,
    transactions
    */
    my_strcat(dest, "{\"blockid\":\"");
    my_strcat(dest, block->blockid);
    my_strcat(dest, "\",\"difficulty\":");
    char diff[100];
    my_strcat(dest, itoa(block->difficulty, diff));
    my_strcat(dest, ",\"prevhash\":\"");
    my_strcat(dest, block->prevhash);
    my_strcat(dest, "\",\"proof\":");
    char noncestr[100];
    my_strcat(dest, itoa(nonce, noncestr));
    my_strcat(dest, ",\"transctions\":");
    my_strcat(dest, block->transactions);
    my_strcat(dest, "}");
    return dest;
}

extern __shared__ char array[];
__global__ void sha256_kernel(char *out_input_string_nonce, unsigned char *out_found_hash, int *out_found, const Block *in_input_string, size_t in_input_string_size, uint8_t difficulty, uint64_t nonce_offset)
{

    // If this is the first thread of the block, init the input block in shared memory
    Block *in;
    if (threadIdx.x == 0)
    {
        memcpy(in, in_input_string, in_input_string_size + 1);
    }

    __syncthreads(); // Ensure the input string has been written in SMEM

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t nonce = idx + nonce_offset;

    // The first byte we can write because there is the input string at the begining
    // Respects the memory padding of 8 bit (char).
    size_t const minArray = static_cast<size_t>(ceil((in_input_string_size + 1) / 8.f) * 8);

    uintptr_t sha_addr = threadIdx.x * (64) + minArray;
    uintptr_t nonce_addr = sha_addr + 32;

    unsigned char *sha = (unsigned char *)&array[sha_addr];
    unsigned char *out = (unsigned char *)&array[nonce_addr];
    memset(out, 0, 32);

    uint8_t size = nonce_to_str(nonce, out);

    assert(size <= 32);

    {
        // unsigned char tmp[32];

        SHA256_CTX ctx;
        sha256_init(&ctx);
        // sha256_update(&ctx, out, size);
        char *dest = (char *)malloc(sizeof(char) * (1));
        dest[0] = '\0';
        //error here (todo: fix)
        sha256_update(&ctx, (unsigned char *)formJSONStr(dest, in, nonce), in_input_string_size);
        free(dest);
        sha256_final(&ctx, sha);

        // Second round of SHA256
        /*sha256_init(&ctx);
        sha256_update(&ctx, tmp, 32);
        sha256_final(&ctx, sha);*/
    }

    // atomicExch(out_found, 1);
    // memcpy(out_found_hash, sha, 32);
    // memcpy(out_input_string_nonce, out, size);
    // memcpy(out_input_string_nonce + size, in, in_input_string_size + 1);

    if (checkZeroPadding(sha, difficulty) && atomicExch(out_found, 1) == 0)
    { // if zero padding, checks if subbing *out_found with 1 is successful
        memcpy(out_found_hash, sha, 32);
        memcpy(out_input_string_nonce, out, size);
        memcpy(out_input_string_nonce + size, in, in_input_string_size + 1);
    }
}

void pre_sha256()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

// Prints a 32 bytes sha256 to the hexadecimal form filled with zeroes
void print_hash(const unsigned char *sha256)
{
    for (uint8_t i = 0; i < 32; ++i)
    {
        std::cout << std::hex << std::setfill('0') << std::setw(2) << static_cast<int>(sha256[i]);
    }
    std::cout << std::dec << std::endl;
}

void print_state()
{
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> last_show_interval = t2 - t_last_updated;

    if (last_show_interval.count() > SHOW_INTERVAL_MS)
    {
        std::chrono::duration<double, std::milli> span = t2 - t_last_updated;
        float ratio = span.count() / 1000;
        std::cout << span.count() << " " << nonce - last_nonce_since_update << std::endl;

        std::cout << std::fixed << static_cast<uint64_t>((nonce - last_nonce_since_update) / ratio) << " hash(es)/s" << std::endl;

        std::cout << std::fixed << "Nonce : " << nonce << std::endl;

        t_last_updated = std::chrono::high_resolution_clock::now();
        last_nonce_since_update = nonce;
    }

    if (*g_found)
    {
        std::cout << g_out << std::endl;
        print_hash(g_hash_out);
    }
}

static Block block;

int main()
{

    hipSetDevice(0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    t_last_updated = std::chrono::high_resolution_clock::now();

    

    std::string blockid;
    std::string prevhash;
    std::string transactions;
    std::cout << "Enter block ID : ";
    getline(std::cin, blockid);
    std::cout << "Enter previous block's hash : ";
    getline(std::cin, prevhash);
    std::cout << "Enter transactions JSON : ";
    getline(std::cin, transactions);
    std::cout << "Enter difficulty : ";
    std::cin >> difficulty;
    std::cout << std::endl;

    strcpy(block.blockid, blockid.c_str());
    strcpy(block.prevhash, prevhash.c_str());
    strcpy(block.transactions, transactions.c_str());
    block.difficulty = difficulty;

    const size_t input_size = sizeof(block);

    // Input string for the device
    char *d_in = nullptr;

    // Create the input string for the device
    hipMalloc(&d_in, input_size + 1);
    hipMemcpy(d_in, &block, input_size + 1, hipMemcpyHostToDevice);

    hipMallocManaged(&g_out, input_size + 32 + 1);
    hipMallocManaged(&g_hash_out, 32);
    hipMallocManaged(&g_found, sizeof(int));
    *g_found = 0;
    std::cout << &g_found << " " << *g_found << " " << g_found << "\n";

    nonce += user_nonce;
    last_nonce_since_update += user_nonce;

    pre_sha256();

    size_t dynamic_shared_size = (ceil((input_size + 1) / 8.f) * 8) + (64 * BLOCK_SIZE);

    std::cout << "Shared memory is " << dynamic_shared_size / 1024 << "KB" << std::endl;

    while (!*g_found)
    {
        //todo: modify to pass block data
        sha256_kernel<<<1, 1, dynamic_shared_size>>>(g_out, g_hash_out, g_found, &block, input_size, difficulty, nonce);

        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
            throw std::runtime_error("Device error");
        }

        nonce += NUMBLOCKS * BLOCK_SIZE;

        print_state();
    }

    std::cout << &g_found << " " << *g_found << " " << g_found << "\n";

    hipFree(g_out);
    hipFree(g_hash_out);
    hipFree(g_found);

    hipFree(d_in);

    hipDeviceReset();

    return 0;
}
